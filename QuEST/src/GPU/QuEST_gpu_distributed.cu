/** @file
 * An implementation of the backend in ../QuEST_internal.h for a Multi-GPU environment.
 * 
 *  @author Jin Bai
*/

# include "QuEST.h"
# include "QuEST_precision.h"
# include "QuEST_validation.h"
# include "QuEST_internal.h"    // purely to resolve getQuESTDefaultSeedKey
# include "mt19937ar.h"

# include <stdlib.h>
# include <stdio.h>
# include <math.h>

# define REDUCE_SHARED_SIZE 512
# define DEBUG 0

// createQuESTEnv

// in single-GPU, 1. GPUExists to check if gpu existed, can potentially extends to return the number of nodes
//                2. Secondly declare an instance of QuESTEnv
//                3. Assign QuESTEnv->rank and QuESTEnv->numRanks
//                4. Assign QuESTEnv->seeds and QuESTEnv->numSeeds, or call seedQuESTDefault()
//                5. Return the QuESTEnv

// in multi-CPU,  1. Declare int variables rank, numRanks, initialized
//                2. Call MPI_Initialized() with address of initialized, see if MPI already initialized
//                3. If MPI not initialized, call MPI_Init, and assign rank and numRanks with MPI_Comm_rank and MPI_Comm_size
//                4. Assign QuESTEnv->rank and QuESTEnv->numRanks
//                5. Call validateNumRanks(), determine if numRanks is power of 2
//                6. Assign QuESTEnv->seeds and QuESTEnv->numSeeds, or call seedQuESTDefault()
//                7. Return the QuESTEnv

// in multi-GPU,  1. Declare int numGPU, rank, numRanks, initialized


int GPUExists(int *numGPU){
    int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess) deviceCount = 0;
    /* machines with no GPUs can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);
        if (properties.major != 9999) { /* 9999 means emulation only */
            ++gpuDeviceCount;
        }
    }
    *numGPU = gpuDeviceCount;  // pass the number of gpu in cuda
    if (gpuDeviceCount) return 1;
    else return 0;
}